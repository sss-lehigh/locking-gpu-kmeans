#include "hip/hip_runtime.h"
#include <fstream>
#include <string>
#include <cstring>
#include <iostream>
#include <sstream>
#include <limits>
#include <chrono>
#include <time.h>
#include "kernels.h"
#include "kmeans.h"

#define M_THREADS 1024
#define U_THREADS 1024
#define R_THREADS 1024
#define MAX_SHMEM 49000

#define GM_CL 1
#define GM_DL 2
#define SM_CL 3
#define SM_DL 4
#define GM_CT 5
#define GM_DT 6
#define SM_CT 7
#define SM_DT 8
#define SC_GCL 9
#define SC_GDL 10

void setFindMembershipParameters(data_t*, int, int&, int&, int&, int&);
void setUpdateParameters(int, data_t*, int, int&, int&, int&, int&);
void setResetParameters(data_t*, int, int&, int&);
int getNumBlocksNeeded(int, int);
std::string getAlgoString(int);

int cuda_kmeans(int algo, int k, data_t* d, int threads) {
        // init clusters...
	float* h_data;
	float* h_clusters;
	float* h_distances;
	int* h_assignments;
	int* h_assignments_prev;
	int* h_nmembers;
	int* h_locks;
	int* h_fg_locks;

	float* data;
	float* clusters;
	float* distances;
	int* assignments;
	int* assignments_prev;
	int* nmembers;
	int* locks;
	int* fg_locks;

	gpuErrchk(hipHostMalloc(&h_data, sizeof(float) * d->numPoints * d->numAttrs));
	gpuErrchk(hipHostMalloc(&h_clusters, sizeof(float) * k * d->numAttrs));
	gpuErrchk(hipHostMalloc(&h_distances, sizeof(float) * d->numPoints));
	gpuErrchk(hipHostMalloc(&h_assignments, sizeof(int) * d->numPoints));
	gpuErrchk(hipHostMalloc(&h_assignments_prev, sizeof(int) * d->numPoints));
	gpuErrchk(hipHostMalloc(&h_nmembers, sizeof(int) * k));
	gpuErrchk(hipHostMalloc(&h_locks, sizeof(int) * k));
	gpuErrchk(hipHostMalloc(&h_fg_locks, sizeof(int) * k * d->numAttrs));

	gpuErrchk(hipMalloc(&data, sizeof(float) * d->numPoints * d->numAttrs));
	gpuErrchk(hipMalloc(&clusters, sizeof(float) * k * d->numAttrs));
	gpuErrchk(hipMalloc(&distances, sizeof(float) * d->numPoints));
	gpuErrchk(hipMalloc(&assignments, sizeof(int) * d->numPoints));
	gpuErrchk(hipMalloc(&assignments_prev, sizeof(int) * d->numPoints));
	gpuErrchk(hipMalloc(&nmembers, sizeof(int) * k));
	gpuErrchk(hipMalloc(&locks, sizeof(int) * k));
	gpuErrchk(hipMalloc(&fg_locks, sizeof(int) * k * d->numAttrs));

//  gpuErrchk(hipMallocManaged(&data, sizeof(float) * d->numPoints * d->numAttrs));
//  gpuErrchk(hipMallocManaged(&clusters, sizeof(float) * k * d->numAttrs));
//  gpuErrchk(hipMallocManaged(&distances, sizeof(float) * d->numPoints));
//  gpuErrchk(hipMallocManaged(&assignments, sizeof(int) * d->numPoints));
//  gpuErrchk(hipMallocManaged(&assignments_prev, sizeof(int) * d->numPoints));
//  gpuErrchk(hipMallocManaged(&nmembers, sizeof(int) * k));
//  gpuErrchk(hipMallocManaged(&locks, sizeof(int) * k));

	for (int i = 0; i < d->numPoints; ++i) {
		for (int j = 0; j < d->numAttrs; ++j) {
			h_data[i * d->numAttrs + j] = d->data[i][j];
		}
	}

	int rand_idx[k];
	setRandomIndices(rand_idx, k);

	for (int i = 0; i < k; ++i) {
		for (int j = 0; j < d->numAttrs; ++j) {
			h_clusters[i * d->numAttrs + j] = d->data[rand_idx[i]][j];
			h_fg_locks[i] = 0;
		}
		h_locks[i] = 0;
		h_nmembers[i] = 0;
	}

	for (int i = 0; i < d->numPoints; ++i) {
		h_distances[i] = std::numeric_limits<float>::max();
		h_assignments[i] = -1;
		h_assignments_prev[i] = -1;
	}

	gpuErrchk(
			hipMemcpyAsync(data, h_data, sizeof(float) * d->numPoints * d->numAttrs,
					hipMemcpyHostToDevice));
	gpuErrchk(
			hipMemcpyAsync(clusters, h_clusters, sizeof(float) * k * d->numAttrs,
					hipMemcpyHostToDevice));
	gpuErrchk(
			hipMemcpyAsync(distances, h_distances, sizeof(float) * d->numPoints,
					hipMemcpyHostToDevice));
	gpuErrchk(
			hipMemcpyAsync(assignments, h_assignments, sizeof(int) * d->numPoints,
					hipMemcpyHostToDevice));
	gpuErrchk(
			hipMemcpyAsync(assignments_prev, h_assignments_prev,
					sizeof(int) * d->numPoints, hipMemcpyHostToDevice));
	gpuErrchk(
			hipMemcpyAsync(nmembers, h_nmembers, sizeof(int) * k,
					hipMemcpyHostToDevice));
	gpuErrchk(
			hipMemcpyAsync(locks, h_locks, sizeof(int) * k, hipMemcpyHostToDevice));
	gpuErrchk(
			hipMemcpyAsync(fg_locks, h_fg_locks, sizeof(int) * k * d->numAttrs, hipMemcpyHostToDevice));
	gpuErrchk(hipDeviceSynchronize());

	// setup threads, blocks and shared memory for find membership kernel
	int m_threads = 0, m_blocks = 0, m_chunk = 0, m_sharedmem = 0;
	setFindMembershipParameters(d, k, m_threads, m_blocks, m_chunk, m_sharedmem);

	// setup threads, blocks and shared memory for the update kernel
	int u_threads = 0, u_blocks = 0, u_chunk = 0, u_sharedmem = 0;
	u_threads = threads;
	setUpdateParameters(algo, d, k, u_threads, u_blocks, u_chunk, u_sharedmem);
	//printf("%d\n", u_chunk);
	//if(u_chunk != 0) printf("%d\n", (k / u_chunk) + 1);

	// setup threads and blocks for the cluster reset kernel
	int r_threads = 0, r_blocks = 0;
	setResetParameters(d, k, r_threads, r_blocks);

	// loop variables
	int loop = 0;
	bool done = false;
	int *update;
	gpuErrchk(hipMallocManaged(&update, sizeof(int))); // accessible on both host and device

	// timing setup
	uint64_t avg_runtime = 0;
	std::chrono::duration<double, std::milli> total_runtime;

	// K-means
	do {
		auto g_start = std::chrono::high_resolution_clock::now();
		*update = 0;

		// assign each datapoint to a cluster
                // find_membership<<<m_blocks, m_threads, m_sharedmem>>>(data, d->numPoints,
                //                 d->numAttrs, clusters, k, m_chunk, assignments, assignments_prev,
                //                 update);
		find_membership_global<<<m_blocks, m_threads>>>(data, d->numPoints,
				d->numAttrs, clusters, k, assignments, assignments_prev,
				update);
		gpuErrchk(hipDeviceSynchronize());

                int reassignments = 0;
                uint64_t iter_runtime = 0;
		gpuErrchk(
				hipMemcpyAsync(h_assignments, assignments, sizeof(int) * d->numPoints,
						hipMemcpyDeviceToHost));
		gpuErrchk(
				hipMemcpyAsync(h_assignments_prev, assignments_prev,
						sizeof(int) * d->numPoints, hipMemcpyDeviceToHost));

		gpuErrchk(hipDeviceSynchronize());
		for(int i = 0; i < d->numPoints; ++i) {
			 if(h_assignments[i] != h_assignments_prev[i]) ++reassignments;
		}

		// check if any were reassigned
		done = true;
		if (*update > 0) {
			done = false;
		}

		// if reassigned, then recalculate clusters
		if (!done) {
			// reset clusters
			reset_clusters<<<r_blocks, r_threads>>>(clusters, nmembers, k,
					d->numAttrs);
			gpuErrchk(hipDeviceSynchronize());

			// constant pointers for kernels
			const float* const_data = data;
			const int* const_ass = assignments;
			const int* const_ass_prev = assignments_prev;
			volatile float* vol_clusters = clusters;
			volatile int* vol_nmembers = nmembers;

			struct timespec cstart, cend;
			// cluster update
			switch (algo) {
			case GM_CL:
                                clock_gettime(CLOCK_REALTIME, &cstart);
				update_clusters_gmcl<<<u_blocks, u_threads>>>(const_data, vol_clusters,
						vol_nmembers, k, d->numPoints, d->numAttrs, const_ass, const_ass_prev,
						locks);
				gpuErrchk(hipDeviceSynchronize())
			        normalize_clusters<<<r_blocks, r_threads>>>(clusters, nmembers, k, d->numAttrs);
				gpuErrchk(hipDeviceSynchronize());
																clock_gettime(CLOCK_REALTIME, &cend);
				break;
			case GM_DL:
                                clock_gettime(CLOCK_REALTIME, &cstart);
				update_clusters_gmdl<<<u_blocks, u_threads>>>(const_data,
						vol_clusters, vol_nmembers, k, d->numPoints, d->numAttrs, const_ass,
						const_ass_prev, fg_locks);
				gpuErrchk(hipDeviceSynchronize())
			        normalize_clusters<<<r_blocks, r_threads>>>(clusters, nmembers, k, d->numAttrs);
				gpuErrchk(hipDeviceSynchronize())
                                clock_gettime(CLOCK_REALTIME, &cend);
				break;
			case SM_CL:
                                clock_gettime(CLOCK_REALTIME, &cstart);
				update_clusters_smcl<<<u_blocks, u_threads, u_sharedmem>>>(
						const_data, clusters, nmembers, k, u_chunk, d->numPoints,
						d->numAttrs, const_ass, const_ass_prev, locks);
				gpuErrchk(hipDeviceSynchronize())
			  normalize_clusters<<<r_blocks, r_threads>>>(clusters, nmembers, k, d->numAttrs);
				gpuErrchk(hipDeviceSynchronize())
                                clock_gettime(CLOCK_REALTIME, &cend);
				break;
			case SM_DL:
                                clock_gettime(CLOCK_REALTIME, &cstart);
				update_clusters_smdl<<<u_blocks, u_threads, u_sharedmem>>>(
						const_data, clusters, nmembers, k, u_chunk, d->numPoints,
						d->numAttrs, const_ass, const_ass_prev, locks);
				gpuErrchk(hipDeviceSynchronize())
			        normalize_clusters<<<r_blocks, r_threads>>>(clusters, nmembers, k, d->numAttrs);
				gpuErrchk(hipDeviceSynchronize())
                                clock_gettime(CLOCK_REALTIME, &cend);
				break;
			case GM_CT:
                                clock_gettime(CLOCK_REALTIME, &cstart);
				update_clusters_gmct<<<u_blocks, u_threads, u_sharedmem>>>(const_data, vol_clusters,
						vol_nmembers, k, d->numPoints, d->numAttrs, const_ass, const_ass_prev,
						locks);
				gpuErrchk(hipDeviceSynchronize())
			        normalize_clusters<<<r_blocks, r_threads>>>(clusters, nmembers, k, d->numAttrs);
				gpuErrchk(hipDeviceSynchronize())
                                clock_gettime(CLOCK_REALTIME, &cend);
				break;
			case GM_DT:
                                clock_gettime(CLOCK_REALTIME, &cstart);
				update_clusters_gmdt<<<u_blocks, u_threads, u_sharedmem>>>(const_data, vol_clusters,
						vol_nmembers, k, d->numPoints, d->numAttrs, const_ass, const_ass_prev,
						locks);
				gpuErrchk(hipDeviceSynchronize())
			  normalize_clusters<<<r_blocks, r_threads>>>(clusters, nmembers, k, d->numAttrs);
				gpuErrchk(hipDeviceSynchronize())
                                clock_gettime(CLOCK_REALTIME, &cend);
				break;
			case SM_CT:
                                clock_gettime(CLOCK_REALTIME, &cstart);
				update_clusters_smct<<<u_blocks, u_threads, u_sharedmem>>>(
						const_data, clusters, nmembers, k, u_chunk, d->numPoints, d->numAttrs, const_ass,
						const_ass_prev, locks);
				gpuErrchk(hipDeviceSynchronize())
			        normalize_clusters<<<r_blocks, r_threads>>>(clusters, nmembers, k, d->numAttrs);
				gpuErrchk(hipDeviceSynchronize())
                                clock_gettime(CLOCK_REALTIME, &cend);
				break;
			case SM_DT:
                                clock_gettime(CLOCK_REALTIME, &cstart);
				update_clusters_smdt<<<u_blocks, u_threads, u_sharedmem>>>(
						const_data, clusters, nmembers, k, u_chunk, d->numPoints, d->numAttrs, const_ass,
						const_ass_prev, locks);
				gpuErrchk(hipDeviceSynchronize())
			  normalize_clusters<<<r_blocks, r_threads>>>(clusters, nmembers, k, d->numAttrs);
				gpuErrchk(hipDeviceSynchronize())
                                clock_gettime(CLOCK_REALTIME, &cend);
				break;
			 case SC_GCL:
        clock_gettime(CLOCK_REALTIME, &cstart);
			 	update_clusters_scgcl<<<u_blocks, u_threads, u_sharedmem>>>(const_data,
			 			clusters, nmembers, k, u_chunk, d->numPoints, d->numAttrs, const_ass,
			 			const_ass_prev, locks);
			 	gpuErrchk(hipDeviceSynchronize())
        clock_gettime(CLOCK_REALTIME, &cend);
			 	break;
			 case SC_GDL:
        clock_gettime(CLOCK_REALTIME, &cstart);
			 	update_clusters_scgdl<<<u_blocks, u_threads, u_sharedmem>>>(const_data,
			 			clusters, nmembers, k, u_chunk, d->numPoints, d->numAttrs, const_ass,
			 		const_ass_prev, fg_locks);
			 	gpuErrchk(hipDeviceSynchronize())
        clock_gettime(CLOCK_REALTIME, &cend);
			 	break;
			}

			// normalize clusters
			gpuErrchk(hipDeviceSynchronize());

			// calculate run times
			auto g_stop = std::chrono::high_resolution_clock::now();
                        iter_runtime = (cend.tv_sec - cstart.tv_sec) * 1000000000 + (cend.tv_nsec - cstart.tv_nsec);
			avg_runtime += iter_runtime; 
			total_runtime += g_stop - g_start;
		}

		// start sanity check
		int total_assigned = 0;
		gpuErrchk(hipMemcpy(h_nmembers, nmembers, sizeof(int) * k, hipMemcpyDeviceToHost));
		gpuErrchk(hipDeviceSynchronize())
		for (int i = 0; i < k; ++i) {
			total_assigned += h_nmembers[i];
		}
		if (total_assigned != d->numPoints) {
			std::cout << "[FAILED]" << std::endl;
			std::cout << "-- loop: " << loop << std::endl;
			std::cout << "-- total assigned: " << total_assigned << std::endl;
			std::cout << "-- number of points: " << d->numPoints << std::endl;
			exit(1);
		}

                //printf("%d\t%lu\n", reassignments, iter_runtime);
	} while (!done && ++loop < 500);

	float tot_rt = total_runtime.count();
	long double updt_rt = ((long double)(avg_runtime) / loop) / 1000000;

//	gpuErrchk(
//			hipMemcpy(h_clusters, clusters, sizeof(float) * k * d->numAttrs,
//					hipMemcpyDeviceToHost))
	std::string algoString = getAlgoString(algo);
	std::printf("%s\t%d\t%f\t%LF\t%d\t%d\t%d\t%d\n", algoString.c_str(), k, tot_rt, updt_rt,
			loop, u_threads, u_blocks, u_sharedmem);
	return 1;
}

void setFindMembershipParameters(data_t* d, int k, int& m_threads, int& m_blocks, int& m_chunk, int& m_sharedmem) {
	m_threads = M_THREADS;
	m_blocks = (
			d->numPoints % m_threads == 0 ?
					d->numPoints / m_threads : d->numPoints / m_threads + 1);
	m_chunk = MAX_SHMEM / (sizeof(float) * d->numAttrs);
	m_sharedmem = sizeof(float) * (d->numAttrs * m_chunk);
}

void setUpdateParameters(int algo, data_t* d, int k, int& u_threads, int& u_blocks, int& u_chunk, int& u_sharedmem){
	int need;
	switch (algo) {
	case GM_CL: // GM-CL
		need = d->numPoints;
		u_blocks = getNumBlocksNeeded(need, u_threads);;
		break;
	case GM_DL: // GM-DL
		need = d->numPoints * d->numAttrs;
		u_blocks = getNumBlocksNeeded(need, u_threads);
		break;
	case SM_CL: // SM-CL
		need = d->numPoints;
		u_blocks = getNumBlocksNeeded(need, u_threads);
		u_chunk = MAX_SHMEM / (sizeof(float) * d->numAttrs + 2 * sizeof(int));
		u_chunk = (u_chunk > k ? k : u_chunk);
		u_sharedmem = sizeof(float) * (d->numAttrs * u_chunk) + 2 * sizeof(int) * u_chunk;
		break;
	case SM_DL: // SM-DL
		need = d->numPoints * d->numAttrs;
		u_blocks = getNumBlocksNeeded(need, u_threads);
		u_chunk =
				MAX_SHMEM
						/ (sizeof(float) * d->numAttrs + sizeof(int)
								+ sizeof(int) * d->numAttrs);
		u_chunk = (u_chunk > k ? k : u_chunk);
		u_sharedmem = sizeof(float) * u_chunk * d->numAttrs + sizeof(int) * u_chunk
				+ sizeof(int) * u_chunk * d->numAttrs;
		break;
	case GM_CT: // GM-CT
		need = k;
		u_blocks = getNumBlocksNeeded(need, u_threads);
		break;
	case GM_DT: // GM-DT
		need = k * d->numAttrs;
		u_blocks = getNumBlocksNeeded(need, u_threads);
		break;
	case SM_CT: // SM-CT
		need = k;
		u_chunk = MAX_SHMEM / (sizeof(float) * d->numAttrs + sizeof(int));
		u_chunk = (u_chunk > u_threads ? u_threads : u_chunk);
		u_threads = u_chunk;
		u_blocks = getNumBlocksNeeded(need, u_threads);
		u_sharedmem = sizeof(float) * (d->numAttrs * u_chunk) + sizeof(int) * u_chunk;
		break;
	case SM_DT: // SM-DT
		need = k * d->numAttrs;
		u_chunk = MAX_SHMEM / (sizeof(float) * d->numAttrs + sizeof(int));
		u_chunk = (u_chunk > u_threads / d->numAttrs ? u_threads / d->numAttrs : u_chunk);
		u_threads = u_chunk * d->numAttrs;
		u_blocks = getNumBlocksNeeded(need, u_threads);
		u_sharedmem = sizeof(float) * (d->numAttrs * u_chunk) + sizeof(int) * u_chunk;
		break;
	case SC_GCL: // SC-GCL
		need = d->numPoints;
		u_blocks = getNumBlocksNeeded(need, u_threads);
		u_chunk = MAX_SHMEM / (sizeof(float) * d->numAttrs + sizeof(int));
		u_sharedmem = sizeof(float) * u_chunk * d->numAttrs + sizeof(int) * u_chunk;
		break;
	case SC_GDL: // SC-GDL
		need = d->numPoints * d->numAttrs;
		u_blocks = getNumBlocksNeeded(need, u_threads);
		u_chunk = MAX_SHMEM / (sizeof(float) * d->numAttrs + sizeof(int));
		u_sharedmem = sizeof(float) * u_chunk * d->numAttrs + sizeof(int) * u_chunk;
		break;
	default:
		std::cout << "Invalid algorithm number provided... exiting." << std::endl;
		exit(1);
	}
}

void setResetParameters(data_t* d, int k, int& r_threads, int& r_blocks) {
	r_threads = R_THREADS;
	r_blocks = getNumBlocksNeeded(k * d->numAttrs, r_threads);
}

int getNumBlocksNeeded(int need, int threads_per_block) {
	return (need % threads_per_block == 0 ? need / threads_per_block : need / threads_per_block + 1);
}

std::string getAlgoString(int algo) {
	switch(algo){
		case GM_CL: return "GM-CL";
		case GM_DL: return "GM-DL";
		case SM_CL: return "SM-CL";
		case SM_DL: return "SM-DL";
		case GM_CT: return "GM-CT";
		case GM_DT: return "GM-DT";
		case SM_CT: return "SM-CT";
		case SM_DT: return "SM-DT";
		case SC_GCL: return "SC-GCL";
		case SC_GDL: return "SC-GDL";
		default: return "";
	}
}
