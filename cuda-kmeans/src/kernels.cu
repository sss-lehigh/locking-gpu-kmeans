#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cfloat>
#include "kernels.h"

//// HELPER KERNELS FOR KMEANS
//////////////////////////////
/*
 * calculates the distance between a datapoint and the provided cluster
 * IN: dataset, clusters
 * OUT: membership 
 * Note: Based on the distance function used in STAMP
 */
__forceinline__ __device__ float calc_distance(
		const float* example, int nfeatures, float* cluster) {
	float dist = 0.0;
	float corr = 0.0;
	for (int i = 0; i < nfeatures; ++i) {
		float y = __fmaf_rd(example[i] - cluster[i], example[i] - cluster[i], corr);
		float t = dist + y;
		corr = y - (t - dist);
		dist = t;
	}
	return sqrtf(dist);
}

/*
 * normalizes the clusters after all thread blocks finish updating centroids
 */
__global__ void normalize_clusters(float* clusters,
		const int* __restrict__ nmembers, const int nclusters,
		const int nfeatures) {
	// find the centroid by getting average
	int p_idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (p_idx < nclusters * nfeatures) {
		int count = nmembers[p_idx / nfeatures];
		if (count < 1) {
			count = 1;
		}
		clusters[p_idx] = clusters[p_idx] / count;
	}
}

/*
 * Reset centroids for new centroid calculation
 */
__global__ void reset_clusters(float* clusters, int* nmembers,
		const int nclusters, const int nfeatures) {
	int p_idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (p_idx < nclusters * nfeatures) {
		clusters[p_idx] = 0.0;
		nmembers[p_idx / nfeatures] = 0;
	}
}
//////////////////////
//////////////////////

//// ASSIGN MEMBERSHIP
//////////////////////
// finds memberhsip for each datapoint using shared memory by copying in clusters and writing clusters 
__global__ void find_membership(const float* data,
		const int npoints, const int nfeatures, float* const clusters,
		const int nclusters, const int cchunk, int* assignments,
		int* assignments_prev, int* update) {
	extern __shared__ float s_clusters[];
	int p_idx = blockIdx.x * blockDim.x + threadIdx.x;

	// in the following
	// cluster_offset = where to start copying clusters
	// nclusters = total number of clusters
	// cchunk = how many clusters can fit in shared memory at once
	float min_dist = FLT_MAX;
	int ass = -1;
	const float limit = 0.99999;
	for (int cluster_offset = 0; cluster_offset < nclusters; cluster_offset +=
			cchunk) {
		// copy in cluster segment 
		if (threadIdx.x < cchunk && cluster_offset + threadIdx.x < nclusters) {
			for (int i = 0; i < nfeatures; ++i) {
				s_clusters[(threadIdx.x * nfeatures) + i] = clusters[(cluster_offset
						+ threadIdx.x) * nfeatures + i];
			}
		}
		__syncthreads();

		// find if new closest cluster
		if (p_idx < npoints) {
			// find assignments for the clusters in shared memory
			for (int i = 0; i < cchunk && cluster_offset + i < nclusters; ++i) {
				//printf("target cluster: %d\n", cluster_offset + i);
				float dist = calc_distance(&data[p_idx * nfeatures], nfeatures,
						&s_clusters[i * nfeatures]);
				if ((dist / min_dist) < limit) {
					min_dist = dist;
					ass = i;
				}
			}
		}
		__syncthreads();
	}

	// if assignment changed then update
	if (p_idx < npoints) {
		if (ass != -1 && ass != assignments[p_idx]) {
			++(*update);
			assignments_prev[p_idx] = assignments[p_idx];
			assignments[p_idx] = ass;
		}
	}
}

// finds membership for each datapoint using global memory only
__global__ void find_membership_global(const float* data,
		const int npoints, const int nfeatures, float* const clusters,
		const int nclusters, int* assignments, int* assignments_prev, int* update) {
	int p_idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (p_idx >= npoints) {
		return;
	}

	float min_dist = FLT_MAX;
	int ass = -1;
	const float limit = 0.99999;
	for (int i = 0; i < nclusters; ++i) {
		float dist = calc_distance(&data[p_idx * nfeatures], nfeatures,
				&clusters[i * nfeatures]);
		if ((dist / min_dist) < limit) {
			min_dist = dist;
			ass = i;
		}
	}

	if (ass != -1 && ass != assignments[p_idx]) {
		++(*update);
		assignments[p_idx] = ass;
	}
	else {
		assignments_prev[p_idx] = assignments[p_idx];
	}
}
/////////////////////////////////
/////////////////////////////////

//// UPDATE CLUSTERS [global locks]
///////////////////////////////////

/*
 * GM-CL
 * updates the centroids based on distance and membership
 * data-centric, meaning each thread is responsible for one datapoint
 * contention between all threads over clusters (only 'k' number can do real work at a time)
 * IN: clusters, nmembers, nclusters, nfeatures, distances, membership, locks
 * OUT: updated clusters
 */
__global__ void update_clusters_gmcl(const float* __restrict__ data,
		volatile float* clusters, volatile int* nmembers, const int nclusters,
		const int npoints, const int nfeatures,
		const int * __restrict__ assignments,
		const int * __restrict__ assignments_prev, int* locks) {
	int p_idx = blockIdx.x * blockDim.x + threadIdx.x;
	volatile int done = 0;

	if (p_idx >= npoints) {
		return;
	} // get rid of unnessesary threads

	// update clusters
	while (!done) {
		int ass = assignments[p_idx];
		if (atomicCAS(&locks[ass], 0, p_idx + 1) == 0) { // +1 needed so that p_idx=0 works
			for (int i = 0; i < nfeatures; ++i) {
				clusters[ass * nfeatures + i] += data[p_idx * nfeatures + i];
			}
			done = 1;
			nmembers[ass] += 1;
			__threadfence();
			atomicExch(&locks[ass], 0);
		}
	}
}

/*
 * GM-DL
 */
__global__ void update_clusters_gmdl(const float* __restrict__ data,
		volatile float* clusters, volatile int* nmembers, const int nclusters,
		const int npoints, const int nfeatures, const int* __restrict__ assignments,
		const int* __restrict__ assignments_prev, int* locks) {
	int p_idx = blockIdx.x * blockDim.x + threadIdx.x;
	volatile int done = 0;

	if (p_idx < nfeatures * npoints) {
		int ass = assignments[p_idx / nfeatures];
		int feature_offset = p_idx % nfeatures;
		int target = ass * nfeatures + feature_offset;

		while (!done) {
			if (atomicCAS(&locks[target], 0, -1) == 0) {
				clusters[target] += data[p_idx];
				if (feature_offset == 0) {
					nmembers[ass] += 1;
				}
				done = 1;
				__threadfence();
				atomicExch(&locks[target], 0);
			}
		}
	}
}
/////////////////////////////////
/////////////////////////////////


//// UPDATE CLUSTERS [shared locks]
///////////////////////////////////

// Intermediate implementation of locks in shared memory
// Each threadblock represents all clusters in shared memory, reduced at the end
// Data partitioned to threads in tb
__global__ void update_clusters_smcl(const float* data,
		volatile float* clusters, volatile int* nmembers, const int nclusters,
		const int cchunk, const int npoints, const int nfeatures,
		const int* assignments,
		const int* assignments_prev, int* locks) {
	int p_idx = blockIdx.x * blockDim.x + threadIdx.x;
	extern __shared__ float s_mem[];

	// pointers to shared memory
	volatile float* s_clusters = reinterpret_cast<volatile float*>(s_mem);
	volatile int* s_nmembers = reinterpret_cast<volatile int*>(s_mem
			+ (cchunk * nfeatures));
	int* s_locks = const_cast<int *>(s_nmembers + cchunk);
	for (int cluster_offset = 0; cluster_offset < nclusters; cluster_offset += cchunk) {
		if(threadIdx.x < cchunk) {
			s_locks[threadIdx.x] = 0;
			s_nmembers[threadIdx.x] = 0;
			for(int i = 0; i < nfeatures; ++i) {
				s_clusters[threadIdx.x * nfeatures + i] = 0.0;
			}
		}
		__syncthreads();

		volatile int done = 0;
		// update cluster for given datapoint
		if (p_idx < npoints) {
			int ass = assignments[p_idx];
			if (ass >= cluster_offset && ass < cluster_offset + cchunk) {
				int s_target = ass % cchunk;
				while (!done) {
					if (atomicCAS(&s_locks[s_target], 0, p_idx + 1) == 0) {
						for (int j = 0; j < nfeatures; ++j) {
							s_clusters[s_target * nfeatures + j] +=
									data[p_idx * nfeatures + j];
						}
						done = 1;
						s_nmembers[s_target] += 1;
						__threadfence();
						atomicExch(&s_locks[s_target], 0);
					}
				}
			}
		}
		__syncthreads(); // needed to ensure s_clusters are completely updated

		int target = cluster_offset + threadIdx.x;
		// use coarse locking to update proper clusters
		if (threadIdx.x < cchunk && target < nclusters) {
			done = 0;
			while (!done) {
				if (atomicCAS(&locks[target], 0, threadIdx.x + 1) == 0) {
					for (int i = 0; i < nfeatures; ++i) {
						clusters[(target) * nfeatures + i] += s_clusters[threadIdx.x
								* nfeatures + i];
					}
					done = 1;
					nmembers[target] += s_nmembers[threadIdx.x];
					__threadfence();
					atomicExch(&locks[target], 0);
				}
			}
		}
		__syncthreads();
	}
}

// More fine grained variation of partitioned data-centric
//
__global__ void update_clusters_smdl(
		const float* data, volatile float* clusters,
		volatile int* nmembers, const int nclusters, const int cchunk,
		const int npoints, const int nfeatures, const int* assignments,
		const int* assignments_prev, int* locks) {
	int p_idx = blockIdx.x * blockDim.x + threadIdx.x;
	extern __shared__ float s_mem[];

	// pointers to shared memory
	volatile float* s_clusters = reinterpret_cast<volatile float*>(s_mem);
	volatile int* s_nmembers = reinterpret_cast<volatile int*>(s_clusters
			+ (cchunk * nfeatures));
	int* s_locks = const_cast<int*>(s_nmembers + cchunk);

	// init shared mem
	for (int cluster_offset = 0; cluster_offset < nclusters; cluster_offset +=
			cchunk) {
		if(threadIdx.x < cchunk) {
			s_nmembers[threadIdx.x] = 0;
			for(int i = 0; i < nfeatures; ++i) {
				s_clusters[threadIdx.x * nfeatures + i] = 0.0;
				s_locks[threadIdx.x * nfeatures + i] = 0;
			}
		}
		__syncthreads();

		volatile int done = 0;
		if (p_idx < npoints * nfeatures) {
			int data_idx = p_idx / nfeatures;
			int ass = assignments[data_idx];
			if (ass >= cluster_offset && ass < cluster_offset + cchunk) {
				int feature_offset = p_idx % nfeatures;
				int s_target = (ass % cchunk) * nfeatures + feature_offset; // feature index into clusters
				int nmem_target = s_target / nfeatures;
				// update cluster for given datapoint feature
				while (!done) {
					if (atomicCAS(&s_locks[s_target], 0, p_idx + 1) == 0) {
						s_clusters[s_target] += data[p_idx];
						if (feature_offset == 0) { // once per datapoint
							s_nmembers[nmem_target] += 1;
						}
						done = 1;
						__threadfence();
						atomicExch(&s_locks[s_target], 0);
					}
				}
			}
		}
		__syncthreads(); // needed to ensure s_clusters are completely updated

		int target = threadIdx.x + cluster_offset;
		if (threadIdx.x < cchunk && target < nclusters) {
			done = 0;
			while (!done) {
				if (atomicCAS(&locks[target], 0, threadIdx.x + 1) == 0) {
					for (int i = 0; i < nfeatures; ++i) {
						clusters[target * nfeatures + i] += s_clusters[threadIdx.x
								* nfeatures + i];
					}
					done = 1;
					nmembers[target] += s_nmembers[threadIdx.x];
					__threadfence();
					atomicExch(&locks[target], 0);
				}
			}
		}
		__syncthreads();
	}
}
//// UPDATE CLUSTERS [no locking]
/////////////////////////////////
__global__ void update_clusters_gmct(const float* data,
		volatile float* clusters, volatile int* nmembers, const int nclusters,
		const int npoints, const int nfeatures,
		const int* assignments,
		const int* assignments_prev, int* locks) {
	extern __shared__ float s_mem[];
	int p_idx = blockDim.x * blockIdx.x + threadIdx.x; // global target cluster
	if (p_idx >= nclusters) {
		return;
	} // if not a real cluster, than exit

	// update centroids
	for (int i = 0; i < npoints; ++i) {
		int ass = assignments[i];
		if (ass == p_idx) {
			for (int j = 0; j < nfeatures; ++j) {
				clusters[p_idx * nfeatures + j] += data[i * nfeatures + j];
			}
			++nmembers[p_idx];
		}
	}
}

__global__ void update_clusters_gmdt(const float* data,
		volatile float* clusters, volatile int* nmembers, const int nclusters,
		const int npoints, const int nfeatures,
		const int* assignments,
		const int* assignments_prev, int* locks) {
	extern __shared__ float s_mem[];
	int p_idx = blockDim.x * blockIdx.x + threadIdx.x; // global target cluster
	if (p_idx >= nclusters * nfeatures) {
		return;
	} // if not a cluster, than exit

	// update centroids
	int dim_offset = p_idx % nfeatures;
	int t_cluster = (int)(p_idx / nfeatures);
	for (int i = 0; i < npoints; ++i) {
		int ass = assignments[i];
		if (ass == t_cluster) {
			clusters[p_idx] += data[i * nfeatures + dim_offset];
			if (dim_offset == 0) {
				++nmembers[t_cluster];
			}
		}
	}
}

/*
 * SM-CT
 */
// Simple shared memory implementation, using one thread for each cluster (mimicks Source{d} implementation, maintains STAMP update approach) [cluster-centric]
__global__ void update_clusters_smct(const float* data,
		volatile float* clusters, volatile int* nmembers, const int nclusters,
		const int cchunk, const int npoints, const int nfeatures,
		const int* assignments,
		const int* assignments_prev, int* locks) {
	extern __shared__ float s_mem[];
	int p_idx = blockDim.x * blockIdx.x + threadIdx.x; // global target cluster
	if (threadIdx.x > cchunk || p_idx >= nclusters) {
		return;
	} // if not a real cluster, than exit

	float* s_clusters = reinterpret_cast<float*>(s_mem);
	int* s_nmembers = reinterpret_cast<int*>(s_clusters + (cchunk * nfeatures));

	int cluster_offset = blockIdx.x * cchunk;
	if (threadIdx.x < cchunk) {
		for (int i = 0; i < nfeatures; ++i) {
			s_clusters[threadIdx.x * nfeatures + i] = 0.0;
		}
		s_nmembers[threadIdx.x] = 0;
	}
	__syncthreads(); // needed to ensure sharedmem init is finished...

	// update centroids
	for (int i = 0; i < npoints; ++i) {
		if (p_idx >= cluster_offset && p_idx < cluster_offset + cchunk) {
			int ass = assignments[i];
			if (ass == p_idx) {
				int s_target = ass % cchunk;
				for (int j = 0; j < nfeatures; ++j) {
					s_clusters[s_target * nfeatures + j] += data[i * nfeatures + j];
				}
				++s_nmembers[s_target];
			}
		}
	}
	__syncthreads();

	// copy out
	if (p_idx >= cluster_offset && p_idx < cluster_offset + cchunk) {
		int s_target = p_idx % cchunk;
		for (int i = 0; i < nfeatures; ++i) {
			clusters[p_idx * nfeatures + i] += s_clusters[s_target * nfeatures + i];
		}
		nmembers[p_idx] = s_nmembers[s_target];
	}
}

__global__ void update_clusters_smdt(const float* data,
		volatile float* clusters, volatile int* nmembers, const int nclusters,
		const int cchunk, const int npoints, const int nfeatures,
		const int* assignments,
		const int* assignments_prev, int* locks) {
	extern __shared__ float s_mem[];
	int p_idx = blockDim.x * blockIdx.x + threadIdx.x; // global target cluster and dimension
	if (p_idx >= nclusters * nfeatures) {
		return;
	} // if not a real cluster, than exit

	float* s_clusters = reinterpret_cast<float*>(s_mem);
	int* s_nmembers = reinterpret_cast<int*>(s_clusters + (cchunk * nfeatures));

	int cluster_offset = blockIdx.x * cchunk;
	if (threadIdx.x < cchunk) {
		for (int i = 0; i < nfeatures; ++i) {
			s_clusters[threadIdx.x * nfeatures + i] = 0.0;
		}
		s_nmembers[threadIdx.x] = 0;
	}
	__syncthreads(); // needed to ensure sharedmem init is finished...

	// update centroids
	int t_cluster = p_idx / nfeatures;
	int dim_offset = p_idx % nfeatures;
	for (int i = 0; i < npoints; ++i) {
		if (t_cluster >= cluster_offset && t_cluster < cluster_offset + cchunk) {
			int ass = assignments[i];
			if (ass == t_cluster) {
				int s_target = t_cluster % cchunk;
				s_clusters[s_target * nfeatures + dim_offset] += data[i * nfeatures + dim_offset];
				if (p_idx % nfeatures == 0) {
					++s_nmembers[s_target];
				}
			}
		}
	}
	__syncthreads();

	// copy out
	if (t_cluster >= cluster_offset && t_cluster < cluster_offset + cchunk) {
		int s_target = t_cluster % cchunk;
		clusters[p_idx] += s_clusters[s_target * nfeatures + dim_offset];
		if (p_idx % nfeatures == 0) {
			nmembers[t_cluster] = s_nmembers[s_target];
		}
	}
}


// threadblock-per-centroid implementation
// Each thread block is assigned a cluster, its threads are then assigned to a feature
// No locking is needed, because there is no overlap between threads (even from different blocks)
// One thread in each tb is responsible for incrementing number of members found for its centroid
// Data is global (can be optimized to be stored read-only)
__global__ void update_clusters_shared_tb(const float* data,
		volatile float* clusters, volatile int* nmembers, const int nclusters,
		const int npoints, const int nfeatures, const int* assignments,
		const int* assignments_prev) {
	extern __shared__ float s_mem[];
	__shared__ int s_nmembers;
	float* s_cluster = reinterpret_cast<float*>(s_mem);

	// ignore unused threads
	if (threadIdx.x >= nfeatures) {
		return;
	}

	// init cluster
	s_cluster[threadIdx.x] = 0.0;
	if (threadIdx.x == 0) {
		s_nmembers = 0;
	}

	// go through all data and update cluster accordingly
	for (int i = 0; i < npoints; ++i) {
		if (assignments[i] == blockIdx.x) {
			s_cluster[threadIdx.x] += data[i * nfeatures + threadIdx.x];
			if (threadIdx.x == 0) {
				++s_nmembers;
			}
		}
	}

	// copy cluster to global memory
	clusters[blockIdx.x * nfeatures + threadIdx.x] = s_cluster[threadIdx.x];
	if (threadIdx.x == 0) {
		nmembers[blockIdx.x] = s_nmembers;
	}
}
/////////////////////////////////
/////////////////////////////////


/////////////////////////////////
/////////////////////////////////

//// UPDATE CLUSTERS [atomics, global]
//////////////////////////////////////
__global__ void update_clusters_atomic(const float* __restrict__ data,
		float* clusters, int* nmembers, const int nclusters, const int npoints,
		const int nfeatures, const int * __restrict__ assignments,
		const int * __restrict__ assignments_prev, int* locks) {
	int p_idx = blockIdx.x * blockDim.x + threadIdx.x;
	volatile int done = 0;

	if (p_idx >= npoints) {
		return;
	} // get rid of unnessesary threads

	// update clusters
	int ass = assignments[p_idx]; // +1 needed so that p_idx=0 works
	for (int i = 0; i < nfeatures; ++i) {
		atomicAdd(&clusters[ass * nfeatures + i], data[p_idx * nfeatures + i]);
	}
	atomicAdd(&nmembers[ass], 1);
}

__global__ void update_clusters_atomic_finegrain(const float* __restrict__ data,
		float* clusters, int* nmembers, const int nclusters, const int npoints,
		const int nfeatures, const int* __restrict__ assignments,
		const int* __restrict__ assignments_prev, int* locks) {
	int p_idx = blockIdx.x * blockDim.x + threadIdx.x;
	volatile int done = 0;

	if (p_idx < nfeatures * npoints) {
		int ass = assignments[p_idx / nfeatures];
		int feature_offset = p_idx % nfeatures;
		int target = ass * nfeatures + feature_offset;

		atomicAdd(&clusters[target], data[p_idx]);
		if (feature_offset == 0) {
			atomicAdd(&nmembers[ass], 1);
		}

	}
}
//////////////////////////////////////
//////////////////////////////////////

__global__ void update_clusters_scgcl(
		const float* data, volatile float* clusters,
		volatile int* nmembers, const int nclusters, const int cchunk,
		const int npoints, const int nfeatures, const int* assignments,
		const int* assignments_prev, int* locks) {

	int p_idx = blockIdx.x * blockDim.x + threadIdx.x;
	extern __shared__ float s_mem[];

	// pointers to shared memory
	volatile float* s_clusters = reinterpret_cast<volatile float*>(s_mem);
	volatile int* s_nmembers = reinterpret_cast<volatile int*>(s_mem
			+ (cchunk * nfeatures));

	for (int cluster_offset = 0; cluster_offset < nclusters; cluster_offset +=
			cchunk) {
		if (threadIdx.x < cchunk) {
			s_nmembers[threadIdx.x] = 0;
			for (int i = 0; i < nfeatures; ++i) {
				s_clusters[threadIdx.x * nfeatures + i] = 0.0;
			}
		}
		__syncthreads(); // needed to ensure sharedmem init is finished...

		volatile int done = 0;
		// update cluster for given datapoint
		if (p_idx < npoints) {
			int ass = assignments[p_idx];
			int s_target = ass % cchunk;
			int c_base = s_target * nfeatures;
			int d_base = p_idx * nfeatures;
			if (ass >= cluster_offset && ass < cluster_offset + cchunk) {
				while (!done) {
					if (atomicCAS(&locks[ass], 0, p_idx + 1) == 0) {
						for (int j = 0; j < nfeatures; ++j) {
							s_clusters[c_base + j] += data[d_base + j];
						}
						done = 1;
						++s_nmembers[s_target];
						atomicExch(&locks[ass], 0);
					}
				}
			}
		}
		__syncthreads(); // needed to ensure s_clusters are completely updated

		// use coarse locking to update proper clusters
		int target = threadIdx.x + cluster_offset;
		if (threadIdx.x < cchunk && target < nclusters) {
			done = 0;
			while (!done) {
				if (atomicCAS(&locks[target], 0, threadIdx.x + 1) == 0) {
					for (int i = 0; i < nfeatures; ++i) {
						clusters[target * nfeatures + i] += s_clusters[threadIdx.x
								* nfeatures + i];
					}
					done = 1;
					nmembers[target] += s_nmembers[threadIdx.x];
					atomicExch(&locks[target], 0);
				}
			}
		}
		__syncthreads();
	}
}

extern __global__ void update_clusters_scgdl(
		const float* __restrict__ data, volatile float* clusters,
		volatile int* nmembers, const int nclusters, const int cchunk,
		const int npoints, const int nfeatures, const int* assignments,
		const int* assignments_prev, int* locks) {
	int p_idx = blockIdx.x * blockDim.x + threadIdx.x;
	extern __shared__ float s_mem[];

	// pointers to shared memory
	volatile float* s_clusters = reinterpret_cast<volatile float*>(s_mem);
	volatile int* s_nmembers = reinterpret_cast<volatile int*>(s_clusters
			+ (cchunk * nfeatures));

	// init shared mem
	for (int cluster_offset = 0; cluster_offset < nclusters; cluster_offset +=
			cchunk) {
		if (threadIdx.x < cchunk) {
			s_nmembers[threadIdx.x] = 0;
			for (int i = 0; i < nfeatures; ++i) {
				s_clusters[threadIdx.x * nfeatures + i] = 0.0;
			}
		}
		__syncthreads(); // needed to ensure sharedmem init is finished...

		int data_idx = p_idx / nfeatures;
		int ass = assignments[data_idx];
		int feature_offset = p_idx % nfeatures;
		int s_target = (ass % cchunk) * nfeatures + feature_offset; // feature index into clusters
		int g_target = ass * nfeatures + feature_offset;
		volatile int done = 0;
		if (p_idx < npoints * nfeatures) {
			if (ass >= cluster_offset && ass < cluster_offset + cchunk) {
				// update cluster for given datapoint feature
				while (!done) {
					if (atomicCAS(&locks[g_target], 0, p_idx + 1) == 0) {
						s_clusters[s_target] += data[p_idx];
						if (feature_offset == 0) { // once per datapoint
							++s_nmembers[s_target / nfeatures];
						}
						done = 1;
						atomicExch(&locks[g_target], 0);
					}
				}
			}
		}
		__syncthreads(); // needed to ensure s_clusters are completely updated

		int target = threadIdx.x + cluster_offset;
		if (threadIdx.x < cchunk && target < nclusters) {
			done = 0;
			while (!done) {
				if (atomicCAS(&locks[target], 0, threadIdx.x + 1) == 0) {
					for (int i = 0; i < nfeatures; ++i) {
						clusters[target * nfeatures + i] += s_clusters[threadIdx.x
								* nfeatures + i];
					}
					done = 1;
					nmembers[target] += s_nmembers[threadIdx.x];
					atomicExch(&locks[target], 0);
				}
			}
		}
		__syncthreads();
	}
}
